/*
 *  gpu_util.cu -- GPU utility functions
 *
 *  Copyright (C) 2010-2013, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2010-2013, Vasileios Karakasis
 */

#include "gpu_util.h"
#include <hip/hip_runtime.h>

void *gpu_alloc(size_t count) {
  void *ret;
  if (hipMalloc(&ret, count) != hipSuccess) {
    ret = NULL;
  }

  return ret;
}

void gpu_free(void *gpuptr) { hipFree(gpuptr); }

int copy_to_gpu(const void *host, void *gpu, size_t count) {
  if (hipMemcpy(gpu, host, count, hipMemcpyHostToDevice) != hipSuccess)
    return -1;
  return 0;
}

int copy_from_gpu(void *host, const void *gpu, size_t count) {
  if (hipMemcpy(host, gpu, count, hipMemcpyDeviceToHost) != hipSuccess)
    return -1;
  return 0;
}

const char *gpu_get_errmsg(hipError_t err) { return hipGetErrorString(err); }

const char *gpu_get_last_errmsg() { return gpu_get_errmsg(hipGetLastError()); }
